#include "hip/hip_runtime.h"
#include "upgma.cuh"
#include <stdio.h>

/**
 * Prints information for each available GPU device on stdout
 */

void printGpuProperties () {
    int nDevices;

    // Store the number of available GPU device in nDevicess
    hipError_t err = hipGetDeviceCount(&nDevices);

    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: hipGetDeviceCount failed!\n");
        exit(1);
    }

    // For each GPU device found, print the information (memory, bandwidth etc.)
    // about the device
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Device memory: %lu\n", prop.totalGlobalMem);
        printf("  Memory Clock Rate (KHz): %d\n",
               prop.memoryClockRate);
        printf("  Memory Bus Width (bits): %d\n",
               prop.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %f\n",
               2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    }
}

uint32_t getIndex(uint32_t numCols, uint32_t i, uint32_t j) {
    return i*numCols + j;
}

UPGMA::ReadDistMat::ReadDistMat(uint32_t size) {
    mat_dim = size;

    hipError_t err;

    distMat    = new uint32_t [mat_dim*mat_dim];
    opMat      = new uint32_t [mat_dim*mat_dim];
    clusterLst = new int [mat_dim*mat_dim];

    err = hipMalloc(&d_distMat, mat_dim*mat_dim*sizeof(uint32_t));
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: hipMalloc failed!\n");
        exit(1);
    }

    err = hipMalloc(&d_opMat, mat_dim*mat_dim*sizeof(uint32_t));
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: hipMalloc failed!\n");
        exit(1);
    }

    err = hipMalloc(&d_clusterLst, mat_dim*mat_dim*sizeof(uint32_t));
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: hipMalloc failed!\n");
        exit(1);
    }
}

void UPGMA::readFile(UPGMA::ReadDistMat* readDistMat) {
    int i = 0;
    int j = 0;
    std::ifstream infile("distMat.txt");
    std::string line;
    while (std::getline(infile, line)) {
        j = 0;
        std::istringstream iss(line);
        std::string token;
        while (std::getline(iss, token, ',')) {
            if (token != "") { // Skip empty tokens
                readDistMat->distMat[getIndex(readDistMat->mat_dim, i, j)] = std::stoi(token);
                j++;
            }
        }
        i++;
    }
}

void UPGMA::transferDistMat(UPGMA::ReadDistMat* readDistMat) {
    hipError_t err;

    uint32_t numReads = readDistMat->mat_dim;

    err = hipMemcpy(readDistMat->d_distMat, readDistMat->distMat, numReads*numReads*sizeof(uint32_t), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: cudaMemCpy failed!\n");
        exit(1);
    }
}

void UPGMA::printDistMat(UPGMA::ReadDistMat* readDistMat) {
    
    uint32_t numReads = readDistMat->mat_dim;
    hipError_t err;
    
    err = hipMemcpy(readDistMat->distMat, readDistMat->d_distMat, numReads*numReads*sizeof(uint32_t), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: cudaMemCpy failed!\n");
        exit(1);
    }

    printf("Distance Matrix:\n");

    for (uint64_t i=0; i < readDistMat->mat_dim; i++) {
        for (uint64_t j=0; j < readDistMat->mat_dim; j++) {
        printf("%u ", readDistMat->distMat[getIndex(readDistMat->mat_dim, i,j)]);
        }
        printf("\n");
    }
}

void UPGMA::printOpMat(UPGMA::ReadDistMat* readDistMat) {
    
    uint32_t numReads = readDistMat->mat_dim;
    hipError_t err;
    
    err = hipMemcpy(readDistMat->opMat, readDistMat->d_opMat, numReads*numReads*sizeof(uint32_t), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: cudaMemCpy failed!\n");
        exit(1);
    }

    printf("Operation Matrix:\n");

    for (uint64_t i=0; i < readDistMat->mat_dim; i++) {
        for (uint64_t j=0; j < readDistMat->mat_dim; j++) {
        printf("%u ", readDistMat->opMat[getIndex(readDistMat->mat_dim, i,j)]);
        }
        printf("\n");
    }
}

void UPGMA::printClusterLst(UPGMA::ReadDistMat* readDistMat) {
    
    uint32_t numReads = readDistMat->mat_dim;
    hipError_t err;
    
    err = hipMemcpy(readDistMat->clusterLst, readDistMat->d_clusterLst, numReads*numReads*sizeof(uint32_t), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: cudaMemCpy failed!\n");
        exit(1);
    }

    printf("Cluster List:\n");

    for (uint64_t i=0; i < readDistMat->mat_dim; i++) {
        for (uint64_t j=0; j < readDistMat->mat_dim; j++) {
        printf("%u ", readDistMat->clusterLst[getIndex(readDistMat->mat_dim, i,j)]);
        }
        printf("\n");
    }
}

void UPGMA::clearDistMat(UPGMA::ReadDistMat* readDistMat) {
    delete[] readDistMat->distMat;
    delete[] readDistMat->opMat;
    delete[] readDistMat->clusterLst;
    
    hipFree(readDistMat->d_distMat);
    hipFree(readDistMat->d_opMat);
    hipFree(readDistMat->d_clusterLst);

    delete readDistMat;
}

__device__ uint32_t getIndexDev(uint32_t numCols, uint32_t i, uint32_t j) {
    return numCols*i + j;
}

__global__ void buildUpgma(
    uint32_t mat_dim,
    uint32_t* d_distMat,
    uint32_t* d_opMat,
    int* d_clusterLst) {
    
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    // int gs = gridDim.x;
    // int bs = blockDim.x;	

    // printf("Starting buildUPGMA kernel\n");
    
    // for (int i = 0; i < 1000; i++){
    //     10+20;
    // }

    // d_distMat[getIndexDev(mat_dim,tx,bx)] += 100;

    if (tx == 0 && bx == 0) {
        //1. mirrorDistMat(MATDIM);
	    //void mirrorDistMat(int size) {
        printf("Starting buildUPGMA kernel\n");
        int *minLoc;
	    int greaterCombLoc;
        for (int itr1_i = 0; itr1_i < mat_dim; itr1_i++) {
            for (int itr1_j = 0; itr1_j < itr1_i; itr1_j++)
                d_distMat[getIndexDev(mat_dim,itr1_j,itr1_i)] = d_distMat[getIndexDev(mat_dim,itr1_i,itr1_j)];
        }
        //}

        //2. initClusterLst(mat_dim);
        //void initClusterLst(int size) {
        for (int itr2_i = 0; itr2_i < mat_dim; itr2_i++) {
            for (int itr2_j = 0; itr2_j < mat_dim; itr2_j++) {
                d_clusterLst[getIndexDev(mat_dim,itr2_i,itr2_j)] = (itr2_j) ? -1 : itr2_i;
            }
        }
        //}

        //initd_opMat(mat_dim);
        //void initd_opMat(int size) {
        for (int itr3_i = 0; itr3_i < mat_dim; itr3_i++) {
            for (int itr3_j = 0; itr3_j < mat_dim; itr3_j++)
                d_opMat[getIndexDev(mat_dim,itr3_i,itr3_j)] = d_distMat[getIndexDev(mat_dim,itr3_i,itr3_j)]*1000;
        }
        //}

        // printd_opMat(mat_dim);

        for (int k = 0; k < mat_dim-2; k++) {
            //minLoc = matMinLoc(mat_dim);
            //int *matMinLoc(int size) {
            int itr4_min = __INT32_MAX__;
            int itr4_min_loc[2];
            for (int itr4_i = 0; itr4_i < mat_dim; itr4_i++) {
                for (int itr4_j = 0; itr4_j < mat_dim; itr4_j++) {
                    if (d_opMat[getIndexDev(mat_dim,itr4_i,itr4_j)] < itr4_min && d_opMat[getIndexDev(mat_dim,itr4_i,itr4_j)] != 0) {
                        itr4_min = d_opMat[getIndexDev(mat_dim,itr4_i,itr4_j)];
                        itr4_min_loc[0] = itr4_i;
                        itr4_min_loc[1] = itr4_j;
                    }
                }
            }
            minLoc = itr4_min_loc;
            //}
            printf("Minx %d, Miny %d\n", minLoc[0], minLoc[1]);
            //greaterCombLoc = arrMax(minLoc, 2);
            //int arrMax(int d[mat_dim], int size) {
            int itr5_arr_max = 0;
            for (int itr5_i = 0; itr5_i < 2; itr5_i++)
                if (minLoc[itr5_i] > itr5_arr_max)
                    itr5_arr_max = minLoc[itr5_i];
            //return itr5_arr_max;
            greaterCombLoc = itr5_arr_max;
            //}

            //grpLeaf(mat_dim, minLoc[0], minLoc[1]);
            //void grpLeaf(int size, int ele1, int ele2) {
            int *itr6_cluster_locs;
            //itr6_cluster_locs = findCluster(mat_dim, minLoc[0], minLoc[1]);
            //int *findCluster(int size, int ele1, int ele2) {
            int itr7_cluster_locs[2];
            int itr7_find_count = 0;
            int itr7_return_flag = 0;
            for (int itr7_i = 0; itr7_i < mat_dim; itr7_i++) {
                for (int itr7_j = 0; itr7_j < mat_dim; itr7_j++) {
                    if (minLoc[0] == d_clusterLst[getIndexDev(mat_dim,itr7_i,itr7_j)]) {
                        itr7_cluster_locs[0] = itr7_i;
                        itr7_find_count++;
                    } else if (minLoc[1] == d_clusterLst[getIndexDev(mat_dim,itr7_i,itr7_j)]) {
                        itr7_cluster_locs[1] = itr7_i;
                        itr7_find_count++;
                    }

                    if (itr7_find_count == 2) {
                        //return itr7_cluster_locs;
                        itr6_cluster_locs = itr7_cluster_locs;
                        itr7_return_flag = 1;
                        break;
                    }
                }
                if(itr7_return_flag == 1) {
                    break;
                }
            }

            //return itr7_cluster_locs;
            if(itr7_return_flag == 0 ) {
                itr6_cluster_locs = itr7_cluster_locs;
            }
            //}

            if (itr6_cluster_locs[0] != itr6_cluster_locs[1]) {
                int itr6_j;
                for (itr6_j = 0; itr6_j < mat_dim; itr6_j++) {
                    if (d_clusterLst[getIndexDev(mat_dim,itr6_cluster_locs[0],itr6_j)] == -1)
                        break;  
                }
                for (int itr6_k = 0; itr6_k < mat_dim; itr6_k++) {
                    int itr6_ele = d_clusterLst[getIndexDev(mat_dim,itr6_cluster_locs[1],itr6_k)];
                    if (itr6_ele == -1)
                        break;
                    d_clusterLst[getIndexDev(mat_dim,itr6_cluster_locs[0],itr6_j)] = itr6_ele;
                    d_clusterLst[getIndexDev(mat_dim,itr6_cluster_locs[1],itr6_k)] = -1;
                    itr6_j++;
                }
            } 
            //}

            for (int i = 0; i < mat_dim; i++) {
                for (int j = 0; j < mat_dim; j++) {
                    if (d_opMat[getIndexDev(mat_dim,i,j)] != 0) {
                        bool itr8_condition0 = false ;
                        bool itr8_condition1 = false ;
                        //int isEleInArr (int minLoc[mat_dim], int 2, int i) {
                        for (int itr8_i = 0; itr8_i < 2; itr8_i++)
                            if (i == minLoc[itr8_i])
                                //return 1;
                                itr8_condition0 = true;
                        //}
                        for (int itr8_i = 0; itr8_i < 2; itr8_i++)
                            if (j == minLoc[itr8_i])
                                //return 1;
                                itr8_condition1 = true;
                        //}
                        //if (isEleInArr(minLoc, 2, i) || isEleInArr(minLoc, 2, j)) {
                        if ( itr8_condition0 || itr8_condition1 ) {
                            if (i == greaterCombLoc || j == greaterCombLoc)
                                d_opMat[getIndexDev(mat_dim,i,j)] = 0;
                            else
                                if (i != j) {
                                    //matEntryUpdate(mat_dim, i, j);
                                    //void matEntryUpdate(int size, int ele1, int ele2) {
                                    int *itr9_cluster_loc;
                                    //itr9_cluster_loc = findCluster(mat_dim, i, j);
                                    //int *findCluster(int size, int ele1, int ele2) {
                                    int itr10_cluster_locs[2];
                                    int itr10_find_count = 0;
                                    int itr10_return_flag = 0;
                                    for (int itr10_i = 0; itr10_i < mat_dim; itr10_i++) {
                                        for (int itr10_j = 0; itr10_j < mat_dim; itr10_j++) {
                                            if (i == d_clusterLst[getIndexDev(mat_dim,itr10_i,itr10_j)]) {
                                                itr10_cluster_locs[0] = itr10_i;
                                                itr10_find_count++;
                                            } else if (j == d_clusterLst[getIndexDev(mat_dim,itr10_i,itr10_j)]) {
                                                itr10_cluster_locs[1] = itr10_i;
                                                itr10_find_count++;
                                            }

                                            if (itr10_find_count == 2) {
                                                //return itr10_cluster_locs;
                                                itr10_return_flag = 1;
                                                itr9_cluster_loc = itr10_cluster_locs;
                                                break;
                                            }
                                        }
                                        if( itr10_return_flag == 1 ) {
                                            break;
                                        }
                                    }

                                    //return itr10_cluster_locs;
                                    if( itr10_return_flag == 0 ) {
                                        itr9_cluster_loc = itr10_cluster_locs;
                                    }
                                    //}
                                    int itr9_distSum = 0;
                                    int itr9_distCount = 0;
                                    int itr9_e1_itr = 0;
                                    int itr9_e2_itr = 0;  

                                    while (d_clusterLst[getIndexDev(mat_dim,itr9_cluster_loc[0],itr9_e1_itr)] != -1) {
                                        itr9_e2_itr = 0;
                                        while (d_clusterLst[getIndexDev(mat_dim,itr9_cluster_loc[1],itr9_e2_itr)] != -1) {
                                            itr9_distSum += d_distMat[getIndexDev(mat_dim, d_clusterLst[getIndexDev(mat_dim, itr9_cluster_loc[0], itr9_e1_itr)], d_clusterLst[getIndexDev(mat_dim, itr9_cluster_loc[1], itr9_e2_itr)])];
                                            itr9_distCount++;
                                            itr9_e2_itr++;
                                        }
                                        itr9_e1_itr++;
                                    }

                                    d_opMat[getIndexDev(mat_dim,i,j)] = itr9_distSum/itr9_distCount;
                                    //}
                                }
                        }
                }
                }
            }
            // printClusterLst(mat_dim);
            // printOpMat(mat_dim);
        }
    }
}

void UPGMA::upgmaBuilder (UPGMA::ReadDistMat* readDistMat) {

    printf("upgmaBuilder invoked\n");
    int numBlocks = 1; // i.e. number of thread blocks on the GPU
    int blockSize = 8;  // i.e. number of GPU threads per thread block

    buildUpgma<<<numBlocks, blockSize>>>(readDistMat->mat_dim, readDistMat->d_distMat, readDistMat->d_opMat, readDistMat->d_clusterLst);    

    hipError_t err;
    
    err = hipMemcpy(readDistMat->distMat, readDistMat->d_distMat, readDistMat->mat_dim*readDistMat->mat_dim*sizeof(uint32_t), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: cudaMemCpy failed!\n");
        exit(1);
    }

    hipDeviceSynchronize();
}