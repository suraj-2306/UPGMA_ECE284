#include "hip/hip_runtime.h"
#include "upgma.cuh"
#include <stdio.h>

/**
 * Prints information for each available GPU device on stdout
 */

void printGpuProperties () {
	int nDevices;

	// Store the number of available GPU device in nDevicess
	hipError_t err = hipGetDeviceCount(&nDevices);

	if (err != hipSuccess) {
		fprintf(stderr, "GPU_ERROR: hipGetDeviceCount failed!\n");
		exit(1);
	}

	// For each GPU device found, print the information (memory, bandwidth etc.)
	// about the device
	for (int i = 0; i < nDevices; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printf("Device Number: %d\n", i);
		printf("  Device name: %s\n", prop.name);
		printf("  Device memory: %lu\n", prop.totalGlobalMem);
		printf("  Memory Clock Rate (KHz): %d\n",
				prop.memoryClockRate);
		printf("  Memory Bus Width (bits): %d\n",
				prop.memoryBusWidth);
		printf("  Peak Memory Bandwidth (GB/s): %f\n",
				2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
	}
}

uint32_t getIndex(uint32_t numCols, uint32_t i, uint32_t j) {
	return i*numCols + j;
}

UPGMA::ReadDistMat::ReadDistMat(uint32_t size) {
	mat_dim = size;

	hipError_t err;

	distMat    = new uint32_t [mat_dim*mat_dim];
	opMat      = new uint32_t [mat_dim*mat_dim];
	clusterLst = new int [mat_dim*mat_dim];

	err = hipMalloc(&d_distMat, mat_dim*mat_dim*sizeof(uint32_t));
	if (err != hipSuccess) {
		fprintf(stderr, "GPU_ERROR: hipMalloc failed!\n");
		exit(1);
	}

	err = hipMalloc(&d_opMat, mat_dim*mat_dim*sizeof(uint32_t));
	if (err != hipSuccess) {
		fprintf(stderr, "GPU_ERROR: hipMalloc failed!\n");
		exit(1);
	}

	err = hipMalloc(&d_clusterLst, mat_dim*mat_dim*sizeof(uint32_t));
	if (err != hipSuccess) {
		fprintf(stderr, "GPU_ERROR: hipMalloc failed!\n");
		exit(1);
	}
}

void UPGMA::readFile(UPGMA::ReadDistMat* readDistMat) {

	printf("Reading from file\n");

	std::string filename = "./../src/serial/distMat.csv";
	std::ifstream file(filename);
	std::string line;    

	if (!file.is_open()) {
		std::cerr << "Error: Unable to open file " << filename << std::endl;
		return;
	}

	int i = 0;
	int j = 0;

	// Read each line of the CSV file
	while (std::getline(file, line)) {
		std::stringstream ss(line);
		int num;
		j = 0;
		// Parse each element in the line
		while (ss >> num) {
			readDistMat->distMat[getIndex(readDistMat->mat_dim, i, j)] = num;
			if (ss.peek() == ',') // Skip the comma
				ss.ignore();
			j++;
		}
		i++;
	}

	file.close();
}

void UPGMA::transferDistMat(UPGMA::ReadDistMat* readDistMat) {
	hipError_t err;

	uint32_t numReads = readDistMat->mat_dim;

	err = hipMemcpy(readDistMat->d_distMat, readDistMat->distMat, numReads*numReads*sizeof(uint32_t), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(stderr, "GPU_ERROR: cudaMemCpy failed!\n");
		exit(1);
	}
}

void UPGMA::printDistMat(UPGMA::ReadDistMat* readDistMat) {

	uint32_t numReads = readDistMat->mat_dim;
	hipError_t err;

	err = hipMemcpy(readDistMat->distMat, readDistMat->d_distMat, numReads*numReads*sizeof(uint32_t), hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		fprintf(stderr, "GPU_ERROR: cudaMemCpy failed!\n");
		exit(1);
	}

	printf("Distance Matrix:\n");

	for (uint64_t i=0; i < readDistMat->mat_dim; i++) {
		for (uint64_t j=0; j < readDistMat->mat_dim; j++) {
			printf("%u ", readDistMat->distMat[getIndex(readDistMat->mat_dim, i,j)]);
		}
		printf("\n");
	}
}

void UPGMA::printOpMat(UPGMA::ReadDistMat* readDistMat) {

	uint32_t numReads = readDistMat->mat_dim;
	hipError_t err;

	err = hipMemcpy(readDistMat->opMat, readDistMat->d_opMat, numReads*numReads*sizeof(uint32_t), hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		fprintf(stderr, "GPU_ERROR: cudaMemCpy failed!\n");
		exit(1);
	}

	printf("Operation Matrix:\n");

	for (uint64_t i=0; i < readDistMat->mat_dim; i++) {
		for (uint64_t j=0; j < readDistMat->mat_dim; j++) {
			printf("%u ", readDistMat->opMat[getIndex(readDistMat->mat_dim, i,j)]);
		}
		printf("\n");
	}
}

void UPGMA::printClusterLst(UPGMA::ReadDistMat* readDistMat) {

	uint32_t numReads = readDistMat->mat_dim;
	hipError_t err;

	err = hipMemcpy(readDistMat->clusterLst, readDistMat->d_clusterLst, numReads*numReads*sizeof(uint32_t), hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		fprintf(stderr, "GPU_ERROR: cudaMemCpy failed!\n");
		exit(1);
	}

	printf("Cluster List:\n");

	for (uint64_t i=0; i < readDistMat->mat_dim; i++) {
		for (uint64_t j=0; j < readDistMat->mat_dim; j++) {
			printf("%u ", readDistMat->clusterLst[getIndex(readDistMat->mat_dim, i,j)]);
		}
		printf("\n");
	}
}

void UPGMA::clearDistMat(UPGMA::ReadDistMat* readDistMat) {
	delete[] readDistMat->distMat;
	delete[] readDistMat->opMat;
	delete[] readDistMat->clusterLst;

	hipFree(readDistMat->d_distMat);
	hipFree(readDistMat->d_opMat);
	hipFree(readDistMat->d_clusterLst);

	delete readDistMat;
}

__device__ uint32_t getIndexDev(uint32_t numCols, uint32_t i, uint32_t j) {
	return numCols*i + j;
}

__global__ void buildUpgma(
		uint32_t mat_dim,
		uint32_t* d_distMat,
		uint32_t* d_opMat,
		int* d_clusterLst) {

	int tx = threadIdx.x;
	int bx = blockIdx.x;
	// int gs = gridDim.x;
	// int bs = blockDim.x;	

	// printf("Starting buildUPGMA kernel\n");

	// for (int i = 0; i < 1000; i++){
	//     10+20;
	// }

	// d_distMat[getIndexDev(mat_dim,tx,bx)] += 100;

//	uint64_t end_condition = batchSize - 1;
//	uint64_t delta = end_condition / gs + 1; // Reads Per Block
//	uint64_t start = (bx)*delta;
//	uint64_t end =(start + delta - 1 > end_condition) ? end_condition : start + delta - 1;

	if(tx==0 && bx==0 ) {

		//1. mirrorDistMat(MATDIM);
		//void mirrorDistMat(int size) {
		printf("Starting buildUPGMA kernel\n");
		int *minLoc;
		int greaterCombLoc;
		for (int itr1_i = 0; itr1_i < mat_dim; itr1_i++) {
			for (int itr1_j = 0; itr1_j < itr1_i; itr1_j++)
				d_distMat[getIndexDev(mat_dim,itr1_j,itr1_i)] = d_distMat[getIndexDev(mat_dim,itr1_i,itr1_j)];
		}
		//}

		//2. initClusterLst(mat_dim);
		//void initClusterLst(int size) {

		//for (int itr2_i = 0; itr2_i < mat_dim; itr2_i++) {
		//	for (int itr2_j = 0; itr2_j < mat_dim; itr2_j++) {
		//		d_clusterLst[getIndexDev(mat_dim,itr2_i,itr2_j)] = (itr2_j) ? -1 : itr2_i;
		//	}
		//}

		for (int itr2_i = 0; itr2_i < mat_dim; itr2_i++) {
			for (int itr2_j = 0; itr2_j < mat_dim; itr2_j++) {
				d_clusterLst[getIndexDev(mat_dim,itr2_i,itr2_j)] = (itr2_j) ? -2 : itr2_i;
			}
			d_clusterLst[getIndexDev(mat_dim,itr2_i,1)] = -1;
		}
		//}

		for (int itr3_i = 0; itr3_i < mat_dim; itr3_i++) {
			for (int itr3_j = 0; itr3_j < mat_dim; itr3_j++)
				//d_opMat[getIndexDev(mat_dim,itr3_i,itr3_j)] = d_distMat[getIndexDev(mat_dim,itr3_i,itr3_j)]*1000;
				d_distMat[getIndexDev(mat_dim,itr3_i,itr3_j)] *=  1000 ;
		}

		//initd_opMat(mat_dim);
		//void initd_opMat(int size) {
		for (int itr3_i = 0; itr3_i < mat_dim; itr3_i++) {
			for (int itr3_j = 0; itr3_j < mat_dim; itr3_j++)
				//d_opMat[getIndexDev(mat_dim,itr3_i,itr3_j)] = d_distMat[getIndexDev(mat_dim,itr3_i,itr3_j)]*1000;
				d_opMat[getIndexDev(mat_dim,itr3_i,itr3_j)] = d_distMat[getIndexDev(mat_dim,itr3_i,itr3_j)];
		}
		//}

		// printd_opMat(mat_dim);

		for (int k = 0; k < mat_dim-2; k++) {
			//minLoc = matMinLoc(mat_dim);
			//int *matMinLoc(int size) {
			int itr4_min = __INT32_MAX__;
			int itr4_min_loc[2];
			for (int itr4_i = 0; itr4_i < mat_dim; itr4_i++) {
				for (int itr4_j = 0; itr4_j < mat_dim; itr4_j++) {
					if (d_opMat[getIndexDev(mat_dim,itr4_i,itr4_j)] < itr4_min && d_opMat[getIndexDev(mat_dim,itr4_i,itr4_j)] != 0) {
						itr4_min = d_opMat[getIndexDev(mat_dim,itr4_i,itr4_j)];
						itr4_min_loc[0] = itr4_i;
						itr4_min_loc[1] = itr4_j;
					}
				}
			}
			minLoc = itr4_min_loc;
			//}
			printf("Minx %d, Miny %d\n", minLoc[0], minLoc[1]);
			//greaterCombLoc = arrMax(minLoc, 2);
			//int arrMax(int d[mat_dim], int size) {
			int itr5_arr_max = 0;
			for (int itr5_i = 0; itr5_i < 2; itr5_i++)
				if (minLoc[itr5_i] > itr5_arr_max)
					itr5_arr_max = minLoc[itr5_i];
			//return itr5_arr_max;
			greaterCombLoc = itr5_arr_max;
			//}

			//grpLeaf(mat_dim, minLoc[0], minLoc[1]);
			//void grpLeaf(int size, int ele1, int ele2) {
			int *itr6_cluster_locs;
			//itr6_cluster_locs = findCluster(mat_dim, minLoc[0], minLoc[1]);
			//int *findCluster(int size, int ele1, int ele2) {
			int itr7_cluster_locs[2];
			//int itr7_find_count = 0;
			//int itr7_return_flag = 0;
			for (int itr7_i = 0; itr7_i < mat_dim; itr7_i++) {
				for (int itr7_j = 0; itr7_j < mat_dim; itr7_j++) {
					if (minLoc[0] == d_clusterLst[getIndexDev(mat_dim,itr7_i,itr7_j)]) {
						itr7_cluster_locs[0] = itr7_i;
						//itr7_find_count++;
					}
					//} else if (minLoc[1] == d_clusterLst[getIndexDev(mat_dim,itr7_i,itr7_j)]) {
					if (minLoc[1] == d_clusterLst[getIndexDev(mat_dim,itr7_i,itr7_j)]) {
						itr7_cluster_locs[1] = itr7_i;
						//itr7_find_count++;
					}

					//if (itr7_find_count == 2) {
					//	//return itr7_cluster_locs;
					//	itr6_cluster_locs = itr7_cluster_locs;
					//	//itr7_return_flag = 1;
					//	//break;
					//}
				}
				//if(itr7_return_flag == 1) {
				//	break;
				//}
			}

			//return itr7_cluster_locs;
			//if(itr7_return_flag == 0 ) {
			//	itr6_cluster_locs = itr7_cluster_locs;
			//}
			//}
			itr6_cluster_locs = itr7_cluster_locs;
			int itr6_j_save = 0;
			if (itr6_cluster_locs[0] != itr6_cluster_locs[1]) {
				int itr6_j;
				//for each thread
				for (itr6_j = 0; itr6_j < mat_dim; itr6_j++) {
					//if (d_clusterLst[getIndexDev(mat_dim,itr6_cluster_locs[0],itr6_j)] == -1)

					// Guard for last iteration? Branch divergence?

					if (d_clusterLst[getIndexDev(mat_dim,itr6_cluster_locs[0],itr6_j)] == -1) // && in the same row)
						//break;  
						itr6_j_save = itr6_j;  
				}
				// syncthread
				// for each thread
				for (int itr6_k = 0; itr6_k < mat_dim; itr6_k++) {
					int itr6_ele = d_clusterLst[getIndexDev(mat_dim,itr6_cluster_locs[1],itr6_k)];
					//if (itr6_ele == -1)
					//	break;
					if (itr6_ele >= -1) {
						d_clusterLst[getIndexDev(mat_dim,itr6_cluster_locs[0],itr6_k+itr6_j_save)] = itr6_ele;
						d_clusterLst[getIndexDev(mat_dim,itr6_cluster_locs[1],itr6_k)] = -2;
					//itr6_j++;
					}
				}
			} 
			//}

			for (int i = 0; i < mat_dim; i++) {
				for (int j = 0; j < mat_dim; j++) {
					if (d_opMat[getIndexDev(mat_dim,i,j)] != 0) {
						bool itr8_condition0 = false ;
						bool itr8_condition1 = false ;
						//int isEleInArr (int minLoc[mat_dim], int 2, int i) {
						for (int itr8_i = 0; itr8_i < 2; itr8_i++)
							if (i == minLoc[itr8_i])
								//return 1;
								itr8_condition0 = true;
						//}
						for (int itr8_i = 0; itr8_i < 2; itr8_i++)
							if (j == minLoc[itr8_i])
								//return 1;
								itr8_condition1 = true;
						//}
						//if (isEleInArr(minLoc, 2, i) || isEleInArr(minLoc, 2, j)) {
						if ( itr8_condition0 || itr8_condition1 ) {
							if (i == greaterCombLoc || j == greaterCombLoc)
								d_opMat[getIndexDev(mat_dim,i,j)] = 0;
							else
								if (i != j) {
									//matEntryUpdate(mat_dim, i, j);
									//void matEntryUpdate(int size, int ele1, int ele2) {
									int *itr9_cluster_loc;
									//itr9_cluster_loc = findCluster(mat_dim, i, j);
									//int *findCluster(int size, int ele1, int ele2) {
									int itr10_cluster_locs[2];
									int itr10_find_count = 0;
									int itr10_return_flag = 0;
									for (int itr10_i = 0; itr10_i < mat_dim; itr10_i++) {
										for (int itr10_j = 0; itr10_j < mat_dim; itr10_j++) {
											if (i == d_clusterLst[getIndexDev(mat_dim,itr10_i,itr10_j)]) {
												itr10_cluster_locs[0] = itr10_i;
												itr10_find_count++;
											} else if (j == d_clusterLst[getIndexDev(mat_dim,itr10_i,itr10_j)]) {
												itr10_cluster_locs[1] = itr10_i;
												itr10_find_count++;
											}

											if (itr10_find_count == 2) {
												//return itr10_cluster_locs;
												itr10_return_flag = 1;
												itr9_cluster_loc = itr10_cluster_locs;
												break;
											}
										}
										if( itr10_return_flag == 1 ) {
											break;
										}
									}

									//return itr10_cluster_locs;
									if( itr10_return_flag == 0 ) {
										itr9_cluster_loc = itr10_cluster_locs;
									}
									//}
									int itr9_distSum = 0;
									int itr9_distCount = 0;
									int itr9_e1_itr = 0;
									int itr9_e2_itr = 0;  

									while (d_clusterLst[getIndexDev(mat_dim,itr9_cluster_loc[0],itr9_e1_itr)] != -1) {
										itr9_e2_itr = 0;
										while (d_clusterLst[getIndexDev(mat_dim,itr9_cluster_loc[1],itr9_e2_itr)] != -1) {
											itr9_distSum += d_distMat[getIndexDev(mat_dim, d_clusterLst[getIndexDev(mat_dim, itr9_cluster_loc[0], itr9_e1_itr)], d_clusterLst[getIndexDev(mat_dim, itr9_cluster_loc[1], itr9_e2_itr)])];
											itr9_distCount++;
											itr9_e2_itr++;
										}
										itr9_e1_itr++;
									}

									d_opMat[getIndexDev(mat_dim,i,j)] = itr9_distSum/itr9_distCount;
									//}
								}
						}
				}
				}
			}
			// printClusterLst(mat_dim);
			// printOpMat(mat_dim);
		}
	}
}

void UPGMA::upgmaBuilder (UPGMA::ReadDistMat* readDistMat) {

	printf("upgmaBuilder invoked\n");
	int numBlocks = 1; // i.e. number of thread blocks on the GPU
	int blockSize = 8;  // i.e. number of GPU threads per thread block

	buildUpgma<<<numBlocks, blockSize>>>(readDistMat->mat_dim, readDistMat->d_distMat, readDistMat->d_opMat, readDistMat->d_clusterLst);    

	hipError_t err;

	err = hipMemcpy(readDistMat->distMat, readDistMat->d_distMat, readDistMat->mat_dim*readDistMat->mat_dim*sizeof(uint32_t), hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		fprintf(stderr, "GPU_ERROR: cudaMemCpy failed!\n");
		exit(1);
	}

	hipDeviceSynchronize();
}
